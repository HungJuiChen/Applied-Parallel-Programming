#include "hip/hip_runtime.h"
#include <cmath>
#include <iostream>
#include "gpu-new-forward.h"
#include <hipblas.h>

#define TILE_WIDTH 16
#define BLOCK_SIZE 256
#define MAX_BATCH_SIZE 1000

__global__ void matrix_unrolling_kernel(const float *input, float *output,
                                        const int Batch, const int Channel,
                                        const int Height, const int Width,
                                        const int K) {
    /*
    Modify this function to implement the input matrix unrolling kernel.

    Function paramter definitions:
    input - input
    output - output
    Batch - batch_size (number of images in x)
    Channel - number of input feature maps
    Height - input height dimension
    Width - input width dimension
    K - kernel height and width (K x K)
    */
    const int Height_out = Height - K + 1;
    const int Width_out = Width - K + 1;

    const int H_unroll = Channel * K * K;
    const int W_unroll = Batch * Height_out * Width_out;

    // Calculate h_unroll and w_unroll using 2D grid and block indices
    int h_unroll = blockIdx.y * blockDim.y + threadIdx.y;
    int w_unroll = blockIdx.x * blockDim.x + threadIdx.x;

    if (h_unroll < H_unroll && w_unroll < W_unroll) {
        int c = h_unroll / (K * K);
        int p = (h_unroll % (K * K)) / K;
        int q = (h_unroll % (K * K)) % K;

        int b = w_unroll / (Height_out * Width_out);
        int remainder = w_unroll % (Height_out * Width_out);
        int h = remainder / Width_out;
        int w = remainder % Width_out;

        int input_row = h + p;
        int input_col = w + q;
        
    
        // We have some nice #defs for you below to simplify indexing. Feel free to use them, or create your own.
        // An example use of these macros:
        // float a = in_4d(0,0,0,0)

        #define in_4d(i3, i2, i1, i0) input[(i3) * (Channel * Height * Width) + (i2) * (Height * Width) + (i1) * (Width) + i0]
        #define out_2d(i1, i0) output[(i1) * W_unroll + (i0)]

        // TODO: Insert your input matrix unrolling kernel code here
        if (input_row < Height && input_col < Width) {
            out_2d(h_unroll, w_unroll) = in_4d(b, c, input_row, input_col);
        } else {
            out_2d(h_unroll, w_unroll) = 0.0f;
        }

        #undef in_4d
        #undef out_2d
    }
}

// Tiled matrix multiplication kernel. Computes C = AB
// You don't need to modify this kernel.
__global__ void matrixMultiplyShared(const float *A, const float *B, float *C,
                                     int numARows, int numAColumns,
                                     int numBRows, int numBColumns,
                                     int numCRows, int numCColumns)
{
    __shared__ float tileA[TILE_WIDTH][TILE_WIDTH];
    __shared__ float tileB[TILE_WIDTH][TILE_WIDTH];

    int by = blockIdx.y, bx = blockIdx.x, ty = threadIdx.y, tx = threadIdx.x;

    int row = by * TILE_WIDTH + ty, col = bx * TILE_WIDTH + tx;
    float val = 0;

    for (int tileId = 0; tileId < (numAColumns - 1) / TILE_WIDTH + 1; tileId++) {
        if (row < numARows && tileId * TILE_WIDTH + tx < numAColumns) {
            tileA[ty][tx] = A[(size_t) row * numAColumns + tileId * TILE_WIDTH + tx];
        } else {
            tileA[ty][tx] = 0;
        }
        if (col < numBColumns && tileId * TILE_WIDTH + ty < numBRows) {
            tileB[ty][tx] = B[((size_t) tileId * TILE_WIDTH + ty) * numBColumns + col];
        } else {
            tileB[ty][tx] = 0;
        }
        __syncthreads();

        if (row < numCRows && col < numCColumns) {
            for (int i = 0; i < TILE_WIDTH; i++) {
                val += tileA[ty][i] * tileB[i][tx];
            }
        }
        __syncthreads();
    }

    if (row < numCRows && col < numCColumns) {
        C[row * numCColumns + col] = val;
    }
}

// Permutes the matmul result.
// The output feature map after matmul is of shape Map_out x Batch x Height_out x Width_out,
// and we need to permute it into Batch x Map_out x Height_out x Width_out.
// You don't need to modify this kernel.
__global__ void matrix_permute_kernel(const float *input, float *output, int Map_out,
                                      int Batch, int image_size) {
    int b = blockIdx.y;
    int x = blockIdx.x * BLOCK_SIZE + threadIdx.x;
    if (x < image_size) {
        for (int m = 0; m < Map_out; m++) {
            output[b * Map_out * image_size + m * image_size + x] =
                    input[m * Batch * image_size + b * image_size + x];
        }
    }
}

__host__ void GPUInterface::conv_forward_gpu_prolog(const float *host_output, const float *host_input, const float *host_mask, float **device_output_ptr, float **device_input_ptr, float **device_mask_ptr, const int Batch, const int Map_out, const int Channel, const int Height, const int Width, const int K)
{
    // TODO: Allocate memory and copy over the relevant data structures to the GPU

    // We pass double pointers for you to initialize the relevant device pointers,
    //  which are passed to the other two functions.

    // Useful snippet for error checking
    // hipError_t error = hipGetLastError();
    // if(error != hipSuccess)
    // {
    //     std::cout<<"CUDA error: "<<hipGetErrorString(error)<<std::endl;
    //     exit(-1);
    // }

    // Allocate device memory for input
    size_t input_size = Batch * Channel * Height * Width * sizeof(float);
    hipMalloc((void**) device_input_ptr, input_size);
    hipMemcpy(*device_input_ptr, host_input, input_size, hipMemcpyHostToDevice);

    // Allocate device memory for output
    const int Height_out = Height - K + 1;
    const int Width_out = Width - K + 1;
    size_t output_size = Batch * Map_out * Height_out * Width_out * sizeof(float);
    hipMalloc((void**) device_output_ptr, output_size);

    // Allocate device memory for mask
    size_t mask_size = Map_out * Channel * K * K * sizeof(float);
    hipMalloc((void**) device_mask_ptr, mask_size);
    hipMemcpy(*device_mask_ptr, host_mask, mask_size, hipMemcpyHostToDevice);

    // Check for errors
    hipError_t error = hipGetLastError();
    if(error != hipSuccess)
    {
        std::cout<<"CUDA error (prolog): "<<hipGetErrorString(error)<<std::endl;
        exit(-1);
    }

}


__host__ void GPUInterface::conv_forward_gpu(float *device_output, const float *device_input, const float *device_mask, const int Batch, const int Map_out, const int Channel, const int Height, const int Width, const int K)
{
    const int Height_out = Height - K + 1;
    const int Width_out = Width - K + 1;
    const int Height_unrolled = Channel * K * K;

    // Initialize cuBLAS
    hipblasHandle_t handle;
    hipblasStatus_t status = hipblasCreate(&handle);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        std::cerr << "Failed to create cuBLAS handle" << std::endl;
        exit(EXIT_FAILURE);
    }

    // Determine the number of mini-batches
    int num_batches = (Batch + MAX_BATCH_SIZE - 1) / MAX_BATCH_SIZE;

    float *unrolled_matrix;  // Pointer to device memory for storing the unrolled matrix
    float *matmul_output;    // Pointer to device memory for storing the result of matrix multiplication
    
    // Allocate device memory for unrolled_matrix and matmul_output for the maximum mini-batch size
    size_t max_unroll_size = Height_unrolled * (MAX_BATCH_SIZE * Height_out * Width_out) * sizeof(float);
    hipMalloc((void**)&unrolled_matrix, max_unroll_size);

    size_t max_matmul_size = Map_out * (MAX_BATCH_SIZE * Height_out * Width_out) * sizeof(float);
    
    hipMalloc((void**)&matmul_output, max_matmul_size);
    // TODO: Set the kernel dimensions and call the matrix unrolling kernel.
    
    // Iterate over each mini-batch
    for(int batch_idx = 0; batch_idx < num_batches; ++batch_idx) {
        // Calculate the current mini-batch size
        int current_batch_size = (batch_idx == num_batches - 1) ? (Batch - batch_idx * MAX_BATCH_SIZE) : MAX_BATCH_SIZE;

        // Calculate current W_unroll
        int current_W_unroll = current_batch_size * Height_out * Width_out;
        // Set the kernel dimensions for unrolling using a 2D grid
        dim3 blockDim_unroll(16, 16);
        dim3 gridDim_unroll((current_W_unroll + blockDim_unroll.x - 1) / blockDim_unroll.x,
                            (Height_unrolled + blockDim_unroll.y - 1) / blockDim_unroll.y);
        
        // Call the matrix unrolling kernel for the current mini-batch
        matrix_unrolling_kernel<<<gridDim_unroll, blockDim_unroll>>>(
            device_input + batch_idx * MAX_BATCH_SIZE * Channel * Height * Width, // Offset input pointer
            unrolled_matrix, 
            current_batch_size, 
            Channel, 
            Height, 
            Width, 
            K
        );

        
        hipError_t error = hipGetLastError();
        if(error != hipSuccess)
        {
            std::cout<<"CUDA error (unrolling kernel): "<<hipGetErrorString(error)<<std::endl;
            exit(-1);
        }

        // Prepare parameters for cuBLAS sgemm
        // A: device_mask (Map_out x (Channel * K * K)) - assuming row-major
        // B: unrolled_matrix ((Channel * K * K) x current_W_unroll) - assuming row-major
        // C: matmul_output (Map_out x current_W_unroll) - to store the result

        // Since cuBLAS is column-major, to perform C = A * B in row-major,
        // we can compute C^T = B^T * A^T

        // Thus, set transa = HIPBLAS_OP_T, transb = HIPBLAS_OP_T
        // and swap A and B in the parameters

        const float alpha = 1.0f;
        const float beta = 0.0f;

        // Dimensions for transposed multiplication
        int lda = Map_out;
        int ldb = Channel * K * K;
        int ldc = Map_out;

        status = hipblasSgemm(handle,
                             HIPBLAS_OP_T, // transa
                             HIPBLAS_OP_T, // transb
                             current_W_unroll, // m: columns of B^T (rows of B)
                             Map_out,          // n: columns of A^T (rows of A)
                             Channel * K * K,  // k: rows of B^T / columns of A^T
                             &alpha,
                             device_mask, lda,     // A: device_mask^T
                             unrolled_matrix, ldb, // B: unrolled_matrix^T
                             &beta,
                             matmul_output, ldc    // C: matmul_output^T
        );

        if (status != HIPBLAS_STATUS_SUCCESS) {
            std::cerr << "cuBLAS sgemm failed" << std::endl;
            exit(EXIT_FAILURE);
        }

        // Permute the result of matrix multiplication
        const int out_image_size = Height_out * Width_out;
        dim3 permute_kernel_grid_dim((out_image_size - 1) / BLOCK_SIZE + 1, current_batch_size, 1);
        matrix_permute_kernel<<<permute_kernel_grid_dim, BLOCK_SIZE>>>(
            matmul_output, 
            device_output + batch_idx * MAX_BATCH_SIZE * Map_out * out_image_size, // Offset output pointer
            Map_out, 
            current_batch_size, 
            out_image_size
        );

        // Check for errors after permutation
        error = hipGetLastError();
        if(error != hipSuccess)
        {
            std::cout<<"CUDA error (permute kernel): "<<hipGetErrorString(error)<<std::endl;
            exit(-1);
        }
    }

    // Destroy cuBLAS handle
    hipblasDestroy(handle);

    hipFree(matmul_output);
    hipFree(unrolled_matrix);
}


__host__ void GPUInterface::conv_forward_gpu_epilog(float *host_output, float *device_output, float *device_input, float *device_mask, const int Batch, const int Map_out, const int Channel, const int Height, const int Width, const int K)
{
    // TODO: Copy the output back to host
    const int Height_out = Height - K + 1;
    const int Width_out = Width - K + 1;
    size_t output_size = Batch * Map_out * Height_out * Width_out * sizeof(float);
    hipMemcpy(host_output, device_output, output_size, hipMemcpyDeviceToHost);
    
    // TODO: Free device memory
    hipFree(device_output);
    hipFree(device_input);
    hipFree(device_mask);

    hipError_t error = hipGetLastError();
    if(error != hipSuccess)
    {
        std::cout<<"CUDA error (epilog): "<<hipGetErrorString(error)<<std::endl;
        exit(-1);
    }
}


__host__ void GPUInterface::get_device_properties()
{
    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    for(int dev = 0; dev < deviceCount; dev++)
    {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, dev);

        std::cout<<"Device "<<dev<<" name: "<<deviceProp.name<<std::endl;
        std::cout<<"Computational capabilities: "<<deviceProp.major<<"."<<deviceProp.minor<<std::endl;
        std::cout<<"Max Global memory size: "<<deviceProp.totalGlobalMem<<std::endl;
        std::cout<<"Max Constant memory size: "<<deviceProp.totalConstMem<<std::endl;
        std::cout<<"Max Shared memory size per block: "<<deviceProp.sharedMemPerBlock<<std::endl;
        std::cout<<"Max threads per block: "<<deviceProp.maxThreadsPerBlock<<std::endl;
        std::cout<<"Max block dimensions: "<<deviceProp.maxThreadsDim[0]<<" x, "<<deviceProp.maxThreadsDim[1]<<" y, "<<deviceProp.maxThreadsDim[2]<<" z"<<std::endl;
        std::cout<<"Max grid dimensions: "<<deviceProp.maxGridSize[0]<<" x, "<<deviceProp.maxGridSize[1]<<" y, "<<deviceProp.maxGridSize[2]<<" z"<<std::endl;
        std::cout<<"Warp Size: "<<deviceProp.warpSize<<std::endl;
    }
}