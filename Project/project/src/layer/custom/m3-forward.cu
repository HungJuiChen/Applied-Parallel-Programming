#include "hip/hip_runtime.h"
#include <cmath>
#include <iostream>
#include "gpu-new-forward.h"

#define TILE_WIDTH 16
#define BLOCK_SIZE 256
#define MAX_BATCH_SIZE 1000

__global__ void fused_conv_kernel(const float *input, const float *mask, float *output,
                                  const int Batch, const int Map_out, const int Channel,
                                  const int Height, const int Width, const int K) {
    const int Height_out = Height - K + 1;
    const int Width_out = Width - K + 1;
    const int H_unroll = Channel * K * K;
    const int W_unroll = Batch * Height_out * Width_out;

    // Shared memory for mask and input tiles
    __shared__ float tileA[TILE_WIDTH][TILE_WIDTH];
    __shared__ float tileB[TILE_WIDTH][TILE_WIDTH];

    int by = blockIdx.y;  // Output feature map index (Map_out dimension)
    int bx = blockIdx.x;  // Column index in the unrolled input matrix
    int ty = threadIdx.y;
    int tx = threadIdx.x;

    int row = by * TILE_WIDTH + ty;  // Index in mask (filter weights)
    int col = bx * TILE_WIDTH + tx;  // Index in unrolled input

    float val = 0.0f;

    // Loop over tiles of the unrolled input
    for (int m = 0; m < (H_unroll - 1) / TILE_WIDTH + 1; ++m) {
        // Load mask tile into shared memory
        if (row < Map_out && m * TILE_WIDTH + tx < H_unroll) {
            tileA[ty][tx] = mask[row * H_unroll + m * TILE_WIDTH + tx];
        } else {
            tileA[ty][tx] = 0.0f;
        }

        // Compute indices for the input
        int h_unroll = m * TILE_WIDTH + ty;
        int w_unroll = col;

        if (h_unroll < H_unroll && w_unroll < W_unroll) {
            int c = h_unroll / (K * K);
            int p = (h_unroll % (K * K)) / K;
            int q = (h_unroll % (K * K)) % K;

            int b = w_unroll / (Height_out * Width_out);
            int remainder = w_unroll % (Height_out * Width_out);
            int h = remainder / Width_out;
            int w = remainder % Width_out;

            int input_row = h + p;
            int input_col = w + q;

            if (b < Batch && c < Channel && input_row < Height && input_col < Width) {
                tileB[ty][tx] = input[b * (Channel * Height * Width) + c * (Height * Width) + input_row * Width + input_col];
            } else {
                tileB[ty][tx] = 0.0f;
            }
        } else {
            tileB[ty][tx] = 0.0f;
        }

        __syncthreads();

        // Perform the multiplication and accumulation
        if (row < Map_out && col < W_unroll) {
            for (int k = 0; k < TILE_WIDTH; ++k) {
                val += tileA[ty][k] * tileB[k][tx];
            }
        }

        __syncthreads();
    }

    // Write the output directly to the correct position
    if (row < Map_out && col < W_unroll) {
        int b = col / (Height_out * Width_out);
        int remainder = col % (Height_out * Width_out);
        int h = remainder / Width_out;
        int w = remainder % Width_out;

        if (b < Batch && h < Height_out && w < Width_out) {
            output[b * (Map_out * Height_out * Width_out) + row * (Height_out * Width_out) + h * Width_out + w] = val;
        }
    }
}


__global__ void matrix_unrolling_kernel(const float *input, float *output,
                                        const int Batch, const int Channel,
                                        const int Height, const int Width,
                                        const int K) {
    /*
    Modify this function to implement the input matrix unrolling kernel.

    Function paramter definitions:
    input - input
    output - output
    Batch - batch_size (number of images in x)
    Channel - number of input feature maps
    Height - input height dimension
    Width - input width dimension
    K - kernel height and width (K x K)
    */
    const int Height_out = Height - K + 1;
    const int Width_out = Width - K + 1;

    const int H_unroll = Channel * K * K;
    const int W_unroll = Batch * Height_out * Width_out;

    // Calculate h_unroll and w_unroll using 2D grid and block indices
    int h_unroll = blockIdx.y * blockDim.y + threadIdx.y;
    int w_unroll = blockIdx.x * blockDim.x + threadIdx.x;

    if (h_unroll < H_unroll && w_unroll < W_unroll) {
        int c = h_unroll / (K * K);
        int p = (h_unroll % (K * K)) / K;
        int q = (h_unroll % (K * K)) % K;

        int b = w_unroll / (Height_out * Width_out);
        int remainder = w_unroll % (Height_out * Width_out);
        int h = remainder / Width_out;
        int w = remainder % Width_out;

        int input_row = h + p;
        int input_col = w + q;
        
    
        // We have some nice #defs for you below to simplify indexing. Feel free to use them, or create your own.
        // An example use of these macros:
        // float a = in_4d(0,0,0,0)

        #define in_4d(i3, i2, i1, i0) input[(i3) * (Channel * Height * Width) + (i2) * (Height * Width) + (i1) * (Width) + i0]
        #define out_2d(i1, i0) output[(i1) * W_unroll + (i0)]

        // TODO: Insert your input matrix unrolling kernel code here
        if (input_row < Height && input_col < Width) {
            out_2d(h_unroll, w_unroll) = in_4d(b, c, input_row, input_col);
        } else {
            out_2d(h_unroll, w_unroll) = 0.0f;
        }

        #undef in_4d
        #undef out_2d
    }
}

// Tiled matrix multiplication kernel. Computes C = AB
// You don't need to modify this kernel.
__global__ void matrixMultiplyShared(const float *A, const float *B, float *C,
                                     int numARows, int numAColumns,
                                     int numBRows, int numBColumns,
                                     int numCRows, int numCColumns)
{
    __shared__ float tileA[TILE_WIDTH][TILE_WIDTH];
    __shared__ float tileB[TILE_WIDTH][TILE_WIDTH];

    int by = blockIdx.y, bx = blockIdx.x, ty = threadIdx.y, tx = threadIdx.x;

    int row = by * TILE_WIDTH + ty, col = bx * TILE_WIDTH + tx;
    float val = 0;

    for (int tileId = 0; tileId < (numAColumns - 1) / TILE_WIDTH + 1; tileId++) {
        if (row < numARows && tileId * TILE_WIDTH + tx < numAColumns) {
            tileA[ty][tx] = A[(size_t) row * numAColumns + tileId * TILE_WIDTH + tx];
        } else {
            tileA[ty][tx] = 0;
        }
        if (col < numBColumns && tileId * TILE_WIDTH + ty < numBRows) {
            tileB[ty][tx] = B[((size_t) tileId * TILE_WIDTH + ty) * numBColumns + col];
        } else {
            tileB[ty][tx] = 0;
        }
        __syncthreads();

        if (row < numCRows && col < numCColumns) {
            for (int i = 0; i < TILE_WIDTH; i++) {
                val += tileA[ty][i] * tileB[i][tx];
            }
        }
        __syncthreads();
    }

    if (row < numCRows && col < numCColumns) {
        C[row * numCColumns + col] = val;
    }
}

// Permutes the matmul result.
// The output feature map after matmul is of shape Map_out x Batch x Height_out x Width_out,
// and we need to permute it into Batch x Map_out x Height_out x Width_out.
// You don't need to modify this kernel.
__global__ void matrix_permute_kernel(const float *input, float *output, int Map_out,
                                      int Batch, int image_size) {
    int b = blockIdx.y;
    int x = blockIdx.x * BLOCK_SIZE + threadIdx.x;
    if (x < image_size) {
        for (int m = 0; m < Map_out; m++) {
            output[b * Map_out * image_size + m * image_size + x] =
                    input[m * Batch * image_size + b * image_size + x];
        }
    }
}

__host__ void GPUInterface::conv_forward_gpu_prolog(const float *host_output, const float *host_input, const float *host_mask, float **device_output_ptr, float **device_input_ptr, float **device_mask_ptr, const int Batch, const int Map_out, const int Channel, const int Height, const int Width, const int K)
{
    // TODO: Allocate memory and copy over the relevant data structures to the GPU

    // Allocate device memory for input
    size_t input_size = Batch * Channel * Height * Width * sizeof(float);
    hipMalloc((void**) device_input_ptr, input_size);
    hipMemcpy(*device_input_ptr, host_input, input_size, hipMemcpyHostToDevice);

    // Allocate device memory for output
    const int Height_out = Height - K + 1;
    const int Width_out = Width - K + 1;
    size_t output_size = Batch * Map_out * Height_out * Width_out * sizeof(float);
    hipMalloc((void**) device_output_ptr, output_size);

    // Allocate device memory for mask
    size_t mask_size = Map_out * Channel * K * K * sizeof(float);
    hipMalloc((void**) device_mask_ptr, mask_size);
    hipMemcpy(*device_mask_ptr, host_mask, mask_size, hipMemcpyHostToDevice);

    // Check for errors
    hipError_t error = hipGetLastError();
    if(error != hipSuccess)
    {
        std::cout<<"CUDA error (prolog): "<<hipGetErrorString(error)<<std::endl;
        exit(-1);
    }

}


__host__ void GPUInterface::conv_forward_gpu(float *device_output, const float *device_input, const float *device_mask, const int Batch, const int Map_out, const int Channel, const int Height, const int Width, const int K)
{
    const int Height_out = Height - K + 1;
    const int Width_out = Width - K + 1;
    const int H_unroll = Channel * K * K;

    // Determine the number of mini-batches
    int num_batches = (Batch + MAX_BATCH_SIZE - 1) / MAX_BATCH_SIZE;

    // Iterate over each mini-batch
    for(int batch_idx = 0; batch_idx < num_batches; ++batch_idx) {
        // Calculate the current mini-batch size
        int current_batch_size = (batch_idx == num_batches - 1) ? (Batch - batch_idx * MAX_BATCH_SIZE) : MAX_BATCH_SIZE;
        int current_W_unroll = current_batch_size * Height_out * Width_out;

        // Set grid and block dimensions
        dim3 dimBlock(TILE_WIDTH, TILE_WIDTH);
        dim3 dimGrid((current_W_unroll + TILE_WIDTH - 1) / TILE_WIDTH,
                     (Map_out + TILE_WIDTH - 1) / TILE_WIDTH);

        // Launch the fused kernel
        fused_conv_kernel<<<dimGrid, dimBlock>>>(
            device_input + batch_idx * MAX_BATCH_SIZE * Channel * Height * Width,
            device_mask,
            device_output + batch_idx * MAX_BATCH_SIZE * Map_out * Height_out * Width_out,
            current_batch_size,
            Map_out,
            Channel,
            Height,
            Width,
            K
        );

        // Error checking
        hipError_t error = hipGetLastError();
        if(error != hipSuccess) {
            std::cout << "CUDA error (fused kernel): " << hipGetErrorString(error) << std::endl;
            exit(-1);
        }
    }
}


__host__ void GPUInterface::conv_forward_gpu_epilog(float *host_output, float *device_output, float *device_input, float *device_mask, const int Batch, const int Map_out, const int Channel, const int Height, const int Width, const int K)
{
    // TODO: Copy the output back to host
    const int Height_out = Height - K + 1;
    const int Width_out = Width - K + 1;
    size_t output_size = Batch * Map_out * Height_out * Width_out * sizeof(float);
    hipMemcpy(host_output, device_output, output_size, hipMemcpyDeviceToHost);
    
    // TODO: Free device memory
    hipFree(device_output);
    hipFree(device_input);
    hipFree(device_mask);

    hipError_t error = hipGetLastError();
    if(error != hipSuccess)
    {
        std::cout<<"CUDA error (epilog): "<<hipGetErrorString(error)<<std::endl;
        exit(-1);
    }
}


__host__ void GPUInterface::get_device_properties()
{
    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    for(int dev = 0; dev < deviceCount; dev++)
    {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, dev);

        std::cout<<"Device "<<dev<<" name: "<<deviceProp.name<<std::endl;
        std::cout<<"Computational capabilities: "<<deviceProp.major<<"."<<deviceProp.minor<<std::endl;
        std::cout<<"Max Global memory size: "<<deviceProp.totalGlobalMem<<std::endl;
        std::cout<<"Max Constant memory size: "<<deviceProp.totalConstMem<<std::endl;
        std::cout<<"Max Shared memory size per block: "<<deviceProp.sharedMemPerBlock<<std::endl;
        std::cout<<"Max threads per block: "<<deviceProp.maxThreadsPerBlock<<std::endl;
        std::cout<<"Max block dimensions: "<<deviceProp.maxThreadsDim[0]<<" x, "<<deviceProp.maxThreadsDim[1]<<" y, "<<deviceProp.maxThreadsDim[2]<<" z"<<std::endl;
        std::cout<<"Max grid dimensions: "<<deviceProp.maxGridSize[0]<<" x, "<<deviceProp.maxGridSize[1]<<" y, "<<deviceProp.maxGridSize[2]<<" z"<<std::endl;
        std::cout<<"Warp Size: "<<deviceProp.warpSize<<std::endl;
    }
}