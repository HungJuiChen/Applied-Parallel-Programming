#include "hip/hip_runtime.h"
#include <cmath>
#include <iostream>
#include "gpu-new-forward.h"

#define TILE_WIDTH 16
#define BLOCK_SIZE 256
#define MAX_BATCH_SIZE 1000

__global__ void matrix_unrolling_kernel(const float *input, float *output,
                                        const int Batch, const int Channel,
                                        const int Height, const int Width,
                                        const int K) {
    /*
    Modify this function to implement the input matrix unrolling kernel.

    Function paramter definitions:
    input - input
    output - output
    Batch - batch_size (number of images in x)
    Channel - number of input feature maps
    Height - input height dimension
    Width - input width dimension
    K - kernel height and width (K x K)
    */
    const int Height_out = Height - K + 1;
    const int Width_out = Width - K + 1;

    const int W_unroll = Batch * Height_out * Width_out;

    // Calculate h_unroll and w_unroll using 2D grid and block indices
    int w_unroll = blockIdx.x * blockDim.x + threadIdx.x;

    if (w_unroll < W_unroll) {
        int b = w_unroll / (Height_out * Width_out);
        int remainder = w_unroll % (Height_out * Width_out);
        int h = remainder / Width_out;
        int w = remainder % Width_out;
        
    
        // We have some nice #defs for you below to simplify indexing. Feel free to use them, or create your own.
        // An example use of these macros:
        // float a = in_4d(0,0,0,0)

        // Unroll over h_unroll
        #pragma unroll
        for (int c = 0; c < Channel; ++c) {
            for (int p = 0; p < K; ++p) {
                for (int q = 0; q < K; ++q) {
                    int h_unroll = c * K * K + p * K + q;
                    int input_row = h + p;
                    int input_col = w + q;

                    float val = 0.0f;
                    if (input_row < Height && input_col < Width) {
                        val = input[b * Channel * Height * Width +
                                    c * Height * Width +
                                    input_row * Width + input_col];
                    }
                    output[h_unroll * W_unroll + w_unroll] = val;
                }
            }
        }
    }
}

// Tiled matrix multiplication kernel. Computes C = AB
// You don't need to modify this kernel.
__global__ void matrixMultiplyShared(const float *A, const float *B, float *C,
                                     int numARows, int numAColumns,
                                     int numBRows, int numBColumns,
                                     int numCRows, int numCColumns)
{
    __shared__ float tileA[TILE_WIDTH][TILE_WIDTH];
    __shared__ float tileB[TILE_WIDTH][TILE_WIDTH];

    int by = blockIdx.y, bx = blockIdx.x, ty = threadIdx.y, tx = threadIdx.x;

    int row = by * TILE_WIDTH + ty, col = bx * TILE_WIDTH + tx;
    float val = 0;

    for (int tileId = 0; tileId < (numAColumns - 1) / TILE_WIDTH + 1; tileId++) {
        if (row < numARows && tileId * TILE_WIDTH + tx < numAColumns) {
            tileA[ty][tx] = A[(size_t) row * numAColumns + tileId * TILE_WIDTH + tx];
        } else {
            tileA[ty][tx] = 0;
        }
        if (col < numBColumns && tileId * TILE_WIDTH + ty < numBRows) {
            tileB[ty][tx] = B[((size_t) tileId * TILE_WIDTH + ty) * numBColumns + col];
        } else {
            tileB[ty][tx] = 0;
        }
        __syncthreads();

        if (row < numCRows && col < numCColumns) {
            // Unrolled loop
            #pragma unroll
            for (int i = 0; i < TILE_WIDTH; i++) {
                val += tileA[ty][i] * tileB[i][tx];
            }  
        }
        __syncthreads();
    }

    if (row < numCRows && col < numCColumns) {
        C[row * numCColumns + col] = val;
    }
}

// Permutes the matmul result.
// The output feature map after matmul is of shape Map_out x Batch x Height_out x Width_out,
// and we need to permute it into Batch x Map_out x Height_out x Width_out.
// You don't need to modify this kernel.
__global__ void matrix_permute_kernel(const float *input, float *output, int Map_out,
                                      int Batch, int image_size) {
    int b = blockIdx.y;
    int x = blockIdx.x * BLOCK_SIZE + threadIdx.x;
    if (x < image_size) {
        // Unroll over Map_out if it's small
        #pragma unroll
        for (int m = 0; m < Map_out; m++) {
            output[b * Map_out * image_size + m * image_size + x] =
                    input[m * Batch * image_size + b * image_size + x];
        }
    }
}

__host__ void GPUInterface::conv_forward_gpu_prolog(const float *host_output, const float *host_input, const float *host_mask, float **device_output_ptr, float **device_input_ptr, float **device_mask_ptr, const int Batch, const int Map_out, const int Channel, const int Height, const int Width, const int K)
{
    // TODO: Allocate memory and copy over the relevant data structures to the GPU

    // Allocate device memory for input
    size_t input_size = Batch * Channel * Height * Width * sizeof(float);
    hipMalloc((void**) device_input_ptr, input_size);
    hipMemcpy(*device_input_ptr, host_input, input_size, hipMemcpyHostToDevice);

    // Allocate device memory for output
    const int Height_out = Height - K + 1;
    const int Width_out = Width - K + 1;
    size_t output_size = Batch * Map_out * Height_out * Width_out * sizeof(float);
    hipMalloc((void**) device_output_ptr, output_size);

    // Allocate device memory for mask
    size_t mask_size = Map_out * Channel * K * K * sizeof(float);
    hipMalloc((void**) device_mask_ptr, mask_size);
    hipMemcpy(*device_mask_ptr, host_mask, mask_size, hipMemcpyHostToDevice);

    // Check for errors
    hipError_t error = hipGetLastError();
    if(error != hipSuccess)
    {
        std::cout<<"CUDA error (prolog): "<<hipGetErrorString(error)<<std::endl;
        exit(-1);
    }

}


__host__ void GPUInterface::conv_forward_gpu(float *device_output, const float *device_input, const float *device_mask, const int Batch, const int Map_out, const int Channel, const int Height, const int Width, const int K)
{
    const int Height_out = Height - K + 1;
    const int Width_out = Width - K + 1;
    const int Height_unrolled = Channel * K * K;

    // Determine the number of mini-batches
    int num_batches = (Batch + MAX_BATCH_SIZE - 1) / MAX_BATCH_SIZE;

    float *unrolled_matrix;  // Pointer to device memory for storing the unrolled matrix
    float *matmul_output;    // Pointer to device memory for storing the result of matrix multiplication
    
    // Allocate device memory for unrolled_matrix and matmul_output for the maximum mini-batch size
    size_t max_unroll_size = Height_unrolled * (MAX_BATCH_SIZE * Height_out * Width_out) * sizeof(float);
    hipMalloc((void**)&unrolled_matrix, max_unroll_size);

    size_t max_matmul_size = Map_out * (MAX_BATCH_SIZE * Height_out * Width_out) * sizeof(float);
    hipMalloc((void**)&matmul_output, max_matmul_size);
    // TODO: Set the kernel dimensions and call the matrix unrolling kernel.
    
    // Iterate over each mini-batch
    for(int batch_idx = 0; batch_idx < num_batches; ++batch_idx) {
        // Calculate the current mini-batch size
        int current_batch_size = (batch_idx == num_batches - 1) ? (Batch - batch_idx * MAX_BATCH_SIZE) : MAX_BATCH_SIZE;

        // Calculate current W_unroll
        int current_W_unroll = current_batch_size * Height_out * Width_out;
        // Set the kernel dimensions for unrolling using a 2D grid
        dim3 blockDim_unroll(16, 16);
        dim3 gridDim_unroll((current_W_unroll + blockDim_unroll.x - 1) / blockDim_unroll.x,
                            (Height_unrolled + blockDim_unroll.y - 1) / blockDim_unroll.y);
        
        // Call the matrix unrolling kernel for the current mini-batch
        matrix_unrolling_kernel<<<gridDim_unroll, blockDim_unroll>>>(
            device_input + batch_idx * MAX_BATCH_SIZE * Channel * Height * Width, // Offset input pointer
            unrolled_matrix, 
            current_batch_size, 
            Channel, 
            Height, 
            Width, 
            K
        );

        
        hipError_t error = hipGetLastError();
        if(error != hipSuccess)
        {
            std::cout<<"CUDA error (unrolling kernel): "<<hipGetErrorString(error)<<std::endl;
            exit(-1);
        }

        // TODO: Set the kernel dimensions and call the matmul kernel
        int numARows = Map_out;
        int numAColumns = Channel * K * K;
        int numBRows = Channel * K * K;
        int numBColumns = current_W_unroll;
        int numCRows = Map_out;
        int numCColumns = current_W_unroll;

        dim3 dimBlock(TILE_WIDTH, TILE_WIDTH);
        dim3 dimGrid((numCColumns - 1)/TILE_WIDTH + 1, (numCRows -1)/TILE_WIDTH + 1);

        // Call the matrix multiplication kernel
        matrixMultiplyShared<<<dimGrid, dimBlock>>>(device_mask, unrolled_matrix, matmul_output,
                                                    numARows, numAColumns,
                                                    numBRows, numBColumns,
                                                    numCRows, numCColumns);

        
        error = hipGetLastError();
        if(error != hipSuccess)
        {
            std::cout<<"CUDA error (matmul kernel): "<<hipGetErrorString(error)<<std::endl;
            exit(-1);
        }

        // Permute the result of matrix multiplication
        const int out_image_size = Height_out * Width_out;
        dim3 permute_kernel_grid_dim((out_image_size - 1) / BLOCK_SIZE + 1, current_batch_size, 1);
        matrix_permute_kernel<<<permute_kernel_grid_dim, BLOCK_SIZE>>>(
            matmul_output, 
            device_output + batch_idx * MAX_BATCH_SIZE * Map_out * out_image_size, // Offset output pointer
            Map_out, 
            current_batch_size, 
            out_image_size
        );

        // Check for errors after permutation
        error = hipGetLastError();
        if(error != hipSuccess)
        {
            std::cout<<"CUDA error (permute kernel): "<<hipGetErrorString(error)<<std::endl;
            exit(-1);
        }
    }

    hipFree(matmul_output);
    hipFree(unrolled_matrix);
}


__host__ void GPUInterface::conv_forward_gpu_epilog(float *host_output, float *device_output, float *device_input, float *device_mask, const int Batch, const int Map_out, const int Channel, const int Height, const int Width, const int K)
{
    // TODO: Copy the output back to host
    const int Height_out = Height - K + 1;
    const int Width_out = Width - K + 1;
    size_t output_size = Batch * Map_out * Height_out * Width_out * sizeof(float);
    hipMemcpy(host_output, device_output, output_size, hipMemcpyDeviceToHost);
    
    // TODO: Free device memory
    hipFree(device_output);
    hipFree(device_input);
    hipFree(device_mask);

    hipError_t error = hipGetLastError();
    if(error != hipSuccess)
    {
        std::cout<<"CUDA error (epilog): "<<hipGetErrorString(error)<<std::endl;
        exit(-1);
    }
}


__host__ void GPUInterface::get_device_properties()
{
    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    for(int dev = 0; dev < deviceCount; dev++)
    {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, dev);

        std::cout<<"Device "<<dev<<" name: "<<deviceProp.name<<std::endl;
        std::cout<<"Computational capabilities: "<<deviceProp.major<<"."<<deviceProp.minor<<std::endl;
        std::cout<<"Max Global memory size: "<<deviceProp.totalGlobalMem<<std::endl;
        std::cout<<"Max Constant memory size: "<<deviceProp.totalConstMem<<std::endl;
        std::cout<<"Max Shared memory size per block: "<<deviceProp.sharedMemPerBlock<<std::endl;
        std::cout<<"Max threads per block: "<<deviceProp.maxThreadsPerBlock<<std::endl;
        std::cout<<"Max block dimensions: "<<deviceProp.maxThreadsDim[0]<<" x, "<<deviceProp.maxThreadsDim[1]<<" y, "<<deviceProp.maxThreadsDim[2]<<" z"<<std::endl;
        std::cout<<"Max grid dimensions: "<<deviceProp.maxGridSize[0]<<" x, "<<deviceProp.maxGridSize[1]<<" y, "<<deviceProp.maxGridSize[2]<<" z"<<std::endl;
        std::cout<<"Warp Size: "<<deviceProp.warpSize<<std::endl;
    }
}