#include "hip/hip_runtime.h"
#include <cmath>
#include <iostream>
#include "gpu-new-forward.h"

#define TILE_WIDTH 16
#define BLOCK_SIZE 256
#define MAX_BATCH_SIZE 1000
#define NUM_STREAMS 4

__global__ void matrix_unrolling_kernel(const float *input, float *output,
                                        const int Batch, const int Channel,
                                        const int Height, const int Width,
                                        const int K) {
    /*
    Modify this function to implement the input matrix unrolling kernel.

    Function paramter definitions:
    input - input
    output - output
    Batch - batch_size (number of images in x)
    Channel - number of input feature maps
    Height - input height dimension
    Width - input width dimension
    K - kernel height and width (K x K)
    */
    const int Height_out = Height - K + 1;
    const int Width_out = Width - K + 1;
    //(void)Height_out; // silence declared but never referenced warning. remove this line when you start working
    //(void)Width_out; // silence declared but never referenced warning. remove this line when you start working

    const int H_unroll = Channel * K * K;
    const int W_unroll = Batch * Height_out * Width_out;

    // Calculate h_unroll and w_unroll using 2D grid and block indices
    int h_unroll = blockIdx.y * blockDim.y + threadIdx.y;
    int w_unroll = blockIdx.x * blockDim.x + threadIdx.x;

    if (h_unroll < H_unroll && w_unroll < W_unroll) {
        int c = h_unroll / (K * K);
        int p = (h_unroll % (K * K)) / K;
        int q = (h_unroll % (K * K)) % K;

        int b = w_unroll / (Height_out * Width_out);
        int remainder = w_unroll % (Height_out * Width_out);
        int h = remainder / Width_out;
        int w = remainder % Width_out;

        int input_row = h + p;
        int input_col = w + q;
        
    
        // We have some nice #defs for you below to simplify indexing. Feel free to use them, or create your own.
        // An example use of these macros:
        // float a = in_4d(0,0,0,0)

        #define in_4d(i3, i2, i1, i0) input[(i3) * (Channel * Height * Width) + (i2) * (Height * Width) + (i1) * (Width) + i0]
        #define out_2d(i1, i0) output[(i1) * W_unroll + (i0)]

        // TODO: Insert your input matrix unrolling kernel code here
        if (input_row < Height && input_col < Width) {
            out_2d(h_unroll, w_unroll) = in_4d(b, c, input_row, input_col);
        } else {
            out_2d(h_unroll, w_unroll) = 0.0f;
        }

        #undef in_4d
        #undef out_2d
    }
}

// Tiled matrix multiplication kernel. Computes C = AB
// You don't need to modify this kernel.
__global__ void matrixMultiplyShared(const float *A, const float *B, float *C,
                                     int numARows, int numAColumns,
                                     int numBRows, int numBColumns,
                                     int numCRows, int numCColumns)
{
    __shared__ float tileA[TILE_WIDTH][TILE_WIDTH];
    __shared__ float tileB[TILE_WIDTH][TILE_WIDTH];

    int by = blockIdx.y, bx = blockIdx.x, ty = threadIdx.y, tx = threadIdx.x;

    int row = by * TILE_WIDTH + ty, col = bx * TILE_WIDTH + tx;
    float val = 0;

    for (int tileId = 0; tileId < (numAColumns - 1) / TILE_WIDTH + 1; tileId++) {
        if (row < numARows && tileId * TILE_WIDTH + tx < numAColumns) {
            tileA[ty][tx] = A[(size_t) row * numAColumns + tileId * TILE_WIDTH + tx];
        } else {
            tileA[ty][tx] = 0;
        }
        if (col < numBColumns && tileId * TILE_WIDTH + ty < numBRows) {
            tileB[ty][tx] = B[((size_t) tileId * TILE_WIDTH + ty) * numBColumns + col];
        } else {
            tileB[ty][tx] = 0;
        }
        __syncthreads();

        if (row < numCRows && col < numCColumns) {
            for (int i = 0; i < TILE_WIDTH; i++) {
                val += tileA[ty][i] * tileB[i][tx];
            }
        }
        __syncthreads();
    }

    if (row < numCRows && col < numCColumns) {
        C[row * numCColumns + col] = val;
    }
}

// Permutes the matmul result.
// The output feature map after matmul is of shape Map_out x Batch x Height_out x Width_out,
// and we need to permute it into Batch x Map_out x Height_out x Width_out.
// You don't need to modify this kernel.
__global__ void matrix_permute_kernel(const float *input, float *output, int Map_out,
                                      int Batch, int image_size) {
    int b = blockIdx.y;
    int x = blockIdx.x * BLOCK_SIZE + threadIdx.x;
    if (x < image_size) {
        for (int m = 0; m < Map_out; m++) {
            output[b * Map_out * image_size + m * image_size + x] =
                    input[m * Batch * image_size + b * image_size + x];
        }
    }
}

__host__ void GPUInterface::conv_forward_gpu_prolog(const float *host_output, const float *host_input, const float *host_mask, float **device_output_ptr, float **device_input_ptr, float **device_mask_ptr, const int Batch, const int Map_out, const int Channel, const int Height, const int Width, const int K)
{
    // TODO: Allocate memory and copy over the relevant data structures to the GPU

    // We pass double pointers for you to initialize the relevant device pointers,
    //  which are passed to the other two functions.

    // Useful snippet for error checking
    // hipError_t error = hipGetLastError();
    // if(error != hipSuccess)
    // {
    //     std::cout<<"CUDA error: "<<hipGetErrorString(error)<<std::endl;
    //     exit(-1);
    // }

    // // Allocate device memory for input
    // size_t input_size = Batch * Channel * Height * Width * sizeof(float);
    // hipMalloc((void**) device_input_ptr, input_size);
    // hipMemcpy(*device_input_ptr, host_input, input_size, hipMemcpyHostToDevice);

    // // Allocate device memory for output
    // const int Height_out = Height - K + 1;
    // const int Width_out = Width - K + 1;
    // size_t output_size = Batch * Map_out * Height_out * Width_out * sizeof(float);
    // hipMalloc((void**) device_output_ptr, output_size);

    // // Allocate device memory for mask
    // size_t mask_size = Map_out * Channel * K * K * sizeof(float);
    // hipMalloc((void**) device_mask_ptr, mask_size);
    // hipMemcpy(*device_mask_ptr, host_mask, mask_size, hipMemcpyHostToDevice);

    // Create CUDA streams
    hipStream_t streams[NUM_STREAMS];
    for(int i = 0; i < NUM_STREAMS; ++i){
        hipStreamCreate(&streams[i]);
    }

    // Allocate memory for input, output, and mask for each stream
    size_t input_size = Batch * Channel * Height * Width * sizeof(float);
    size_t output_size = Batch * Map_out * (Height - K + 1) * (Width - K + 1) * sizeof(float);
    size_t mask_size = Map_out * Channel * K * K * sizeof(float);

    // Allocate device memory asynchronously for each stream
    for(int i = 0; i < NUM_STREAMS; ++i){
        hipMallocAsync(&device_input_ptr[i], input_size / NUM_STREAMS, streams[i]);
        hipMallocAsync(&device_output_ptr[i], output_size / NUM_STREAMS, streams[i]);
        hipMallocAsync(&device_mask_ptr[i], mask_size, streams[i]);

        // Asynchronously copy mask since it's common across streams
        hipMemcpyAsync(device_mask_ptr[i], host_mask, mask_size, hipMemcpyHostToDevice, streams[i]);
    }

    // Asynchronously copy input data in chunks per stream
    for(int i = 0; i < NUM_STREAMS; ++i){
        size_t offset = (input_size / NUM_STREAMS) * i;
        hipMemcpyAsync(device_input_ptr[i], host_input + offset / sizeof(float), input_size / NUM_STREAMS, hipMemcpyHostToDevice, streams[i]);
    }

    // Check for errors
    hipError_t error = hipGetLastError();
    if(error != hipSuccess)
    {
        std::cout<<"CUDA error (prolog): "<<hipGetErrorString(error)<<std::endl;
        exit(-1);
    }

}


__host__ void GPUInterface::conv_forward_gpu(float *device_output[], const float *device_input[], const float *device_mask[], const int Batch, const int Map_out, const int Channel, const int Height, const int Width, const int K)
{
    const int Height_out = Height - K + 1;
    const int Width_out = Width - K + 1;
    const int Height_unrolled = Channel * K * K;

    // Determine the number of mini-batches
    // int num_batches = (Batch + MAX_BATCH_SIZE - 1) / MAX_BATCH_SIZE;

    // Determine the number of mini-batches per stream
    int batches_per_stream = (Batch + NUM_STREAMS - 1) / NUM_STREAMS;

    float *unrolled_matrix[NUM_STREAMS];  // Pointer to device memory for storing the unrolled matrix
    float *matmul_output[NUM_STREAMS];    // Pointer to device memory for storing the result of matrix multiplication
    
    // Allocate device memory for unrolled_matrix and matmul_output for the maximum mini-batch size
    // size_t max_unroll_size = Height_unrolled * (MAX_BATCH_SIZE * Height_out * Width_out) * sizeof(float);
    // hipMalloc((void**)&unrolled_matrix, max_unroll_size);

    // size_t max_matmul_size = Map_out * (MAX_BATCH_SIZE * Height_out * Width_out) * sizeof(float);
    
    // hipMalloc((void**)&matmul_output, max_matmul_size);

    for(int i = 0; i < NUM_STREAMS; ++i){
        size_t current_batch_size = (i < Batch % NUM_STREAMS) ? (batches_per_stream) : (batches_per_stream - 1);
        size_t current_W_unroll = current_batch_size * Height_out * Width_out;

        size_t unroll_size = Height_unrolled * current_W_unroll * sizeof(float);
        size_t matmul_size = Map_out * current_W_unroll * sizeof(float);

        hipMallocAsync(&unrolled_matrix[i], unroll_size, streams[i]);
        hipMallocAsync(&matmul_output[i], matmul_size, streams[i]);
    }
    // TODO: Set the kernel dimensions and call the matrix unrolling kernel.
    
    // Iterate over each mini-batch
    // for(int batch_idx = 0; batch_idx < num_batches; ++batch_idx) {
    for(int i = 0; i < NUM_STREAMS; ++i){
        // Calculate the current mini-batch size
        size_t current_batch_size = (i < Batch % NUM_STREAMS) ? (batches_per_stream) : (batches_per_stream - 1);
        if(current_batch_size <= 0) continue;

        // Calculate current W_unroll
        size_t current_W_unroll = current_batch_size * Height_out * Width_out;
        // Set the kernel dimensions for unrolling using a 2D grid
        dim3 blockDim_unroll(16, 16);
        dim3 gridDim_unroll((current_W_unroll + blockDim_unroll.x - 1) / blockDim_unroll.x,
                            (Height_unrolled + blockDim_unroll.y - 1) / blockDim_unroll.y);
        
        // Call the matrix unrolling kernel for the current mini-batch
        matrix_unrolling_kernel<<<gridDim_unroll, blockDim_unroll, 0, streams[i]>>>(
            device_input[i] + batch_idx * MAX_BATCH_SIZE * Channel * Height * Width, // Offset input pointer
            unrolled_matrix[i], 
            current_batch_size, 
            Channel, 
            Height, 
            Width, 
            K
        );

        
        hipError_t error = hipGetLastError();
        if(error != hipSuccess)
        {
            std::cout<<"CUDA error (unrolling kernel): "<<hipGetErrorString(error)<<std::endl;
            exit(-1);
        }

        // TODO: Set the kernel dimensions and call the matmul kernel
        int numARows = Map_out;
        int numAColumns = Channel * K * K;
        int numBRows = Channel * K * K;
        int numBColumns = current_W_unroll;
        int numCRows = Map_out;
        int numCColumns = current_W_unroll;

        dim3 dimBlock(TILE_WIDTH, TILE_WIDTH);
        dim3 dimGrid((numCColumns - 1)/TILE_WIDTH + 1, (numCRows -1)/TILE_WIDTH + 1);

        // Call the matrix multiplication kernel
        matrixMultiplyShared<<<dimGrid, dimBlock, 0, streams[i]>>>(device_mask[i], unrolled_matrix[i], matmul_output[i],
                                                    numARows, numAColumns,
                                                    numBRows, numBColumns,
                                                    numCRows, numCColumns);

        
        error = hipGetLastError();
        if(error != hipSuccess)
        {
            std::cout<<"CUDA error (matmul kernel): "<<hipGetErrorString(error)<<std::endl;
            exit(-1);
        }

        // Permute the result of matrix multiplication
        const int out_image_size = Height_out * Width_out;
        dim3 permute_kernel_grid_dim((out_image_size - 1) / BLOCK_SIZE + 1, current_batch_size, 1);
        matrix_permute_kernel<<<permute_kernel_grid_dim, BLOCK_SIZE, 0, streams[i]>>>(
            matmul_output[i], 
            device_output[i] + batch_idx * MAX_BATCH_SIZE * Map_out * out_image_size, // Offset output pointer
            Map_out, 
            current_batch_size, 
            out_image_size
        );

        // Check for errors after permutation
        error = hipGetLastError();
        if(error != hipSuccess)
        {
            std::cout<<"CUDA error (permute kernel): "<<hipGetErrorString(error)<<std::endl;
            exit(-1);
        }
    }

    // Synchronize all streams to ensure completion
    for(int i = 0; i < NUM_STREAMS; ++i){
        hipStreamSynchronize(streams[i]);
        hipFreeAsync(unrolled_matrix[i], streams[i]);
        hipFreeAsync(matmul_output[i], streams[i]);
        hipStreamDestroy(streams[i]);
    }

    // Check for any remaining errors
    hipError_t final_error = hipGetLastError();
    if(final_error != hipSuccess)
    {
        std::cout << "CUDA error (gpu): " << hipGetErrorString(final_error) << std::endl;
        exit(-1);
    }
}


__host__ void GPUInterface::conv_forward_gpu_epilog(float *host_output, float *device_output[], float *device_input[], float *device_mask[], const int Batch, const int Map_out, const int Channel, const int Height, const int Width, const int K)
{
    // TODO: Copy the output back to host
    const int Height_out = Height - K + 1;
    const int Width_out = Width - K + 1;
    size_t output_size = Batch * Map_out * Height_out * Width_out * sizeof(float)/ NUM_STREAMS;
    //hipMemcpy(host_output, device_output, output_size, hipMemcpyDeviceToHost);
    
    
    // Asynchronously copy the output back to host for each stream
    for(int i = 0; i < NUM_STREAMS; ++i){
        size_t offset = (Batch / NUM_STREAMS) * i * Map_out * Height_out * Width_out;
        hipMemcpyAsync(
            host_output + offset / sizeof(float), 
            device_output[i], 
            output_size, 
            hipMemcpyDeviceToHost, 
            streams[i]
        );
    }
    // TODO: Free device memory
    // Synchronize all streams to ensure all copies are complete
    for(int i = 0; i < NUM_STREAMS; ++i){
        hipStreamSynchronize(streams[i]);
        hipFreeAsync(device_output[i], streams[i]);
        hipFreeAsync(device_input[i], streams[i]);
        hipFreeAsync(device_mask[i], streams[i]);
        hipStreamDestroy(streams[i]);
    }

    hipError_t error = hipGetLastError();
    if(error != hipSuccess)
    {
        std::cout<<"CUDA error (epilog): "<<hipGetErrorString(error)<<std::endl;
        exit(-1);
    }
}


__host__ void GPUInterface::get_device_properties()
{
    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    for(int dev = 0; dev < deviceCount; dev++)
    {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, dev);

        std::cout<<"Device "<<dev<<" name: "<<deviceProp.name<<std::endl;
        std::cout<<"Computational capabilities: "<<deviceProp.major<<"."<<deviceProp.minor<<std::endl;
        std::cout<<"Max Global memory size: "<<deviceProp.totalGlobalMem<<std::endl;
        std::cout<<"Max Constant memory size: "<<deviceProp.totalConstMem<<std::endl;
        std::cout<<"Max Shared memory size per block: "<<deviceProp.sharedMemPerBlock<<std::endl;
        std::cout<<"Max threads per block: "<<deviceProp.maxThreadsPerBlock<<std::endl;
        std::cout<<"Max block dimensions: "<<deviceProp.maxThreadsDim[0]<<" x, "<<deviceProp.maxThreadsDim[1]<<" y, "<<deviceProp.maxThreadsDim[2]<<" z"<<std::endl;
        std::cout<<"Max grid dimensions: "<<deviceProp.maxGridSize[0]<<" x, "<<deviceProp.maxGridSize[1]<<" y, "<<deviceProp.maxGridSize[2]<<" z"<<std::endl;
        std::cout<<"Warp Size: "<<deviceProp.warpSize<<std::endl;
    }
}